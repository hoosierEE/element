#include "hip/hip_runtime.h"
// Perhaps a crazy idea.
// Compiler and interpreter for a K dialect, implemented in cuda C++.
//
// limitations:
//   line length must be < 256
//
// compile:
//   $ nvcc scanner.cu
//
// run a script file:
//   $ ./a.out < filename.ext
//
// eval a string:
//   $ ./a.out "a:7; a*2+3"  # prints 35
//
// start interactive interpreter:
//   $ ./a.out  # start interpreter
#include <iostream>

// ascii [32-127) (note space at start)
//  !"#$%&'()*+,-./0123456789:;<=>?@ABCDEFGHIJKLMNOPQRSTUVWXYZ[\]^_`abcdefghijklmnopqrstuvwxyz{|}~
// wvqvvvvappvvvvavnnnnnnnnnnvpvvvvvnnnnnnnnnnnnnnnnnnnnnnnnnnpvpvvpnnnnnnnnnnnnnnnnnnnnnnnnnnpvpv
// w______________________________________________________________________________________________ whitespace
// __q____________________________________________________________________________________________ quote
// _______a______a________________________________________________________________________________ adverb
// ________________nnnnnnnnnn_______nnnnnnnnnnnnnnnnnnnnnnnnnn______nnnnnnnnnnnnnnnnnnnnnnnnnn____ noun
// ________pp_________________p_______________________________p_p__p__________________________p_p_ punctuation
// _v_vvvv___vvvv_v__________v_vvvvv___________________________v_vv____________________________v_v verb
// A typical CPU scanner is sequential and stateful but we use a different approach here.
// This one breaks scanning into two phases:
// 1. context-free classify each character
// 2. add contextual info to refine class for each char, resulting in tokens
// The scan function below does step 1.

constexpr int LINE_LEN{256};
enum CharClass {END,SPACE,QUOTE,ADVERB,ALPHA,NUMBER,SYMBOL,LP,RP,LS,RS,LC,RC,COLON,DOT,SEM,VERB,UNK};

__global__ void scan(const char *x, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ int x[LINE_LEN];
  if (i < n) {
    char c=x[i];
    char d=c|' '; // tolower
    if      (c == ' ')               x[i] = SPACE;
    else if (c == '"')               x[i] = QUOTE;
    else if (c == '\'' or c == '.')  x[i] = ADVERB;
    else if (d >= 'a' and d <= 'z')  x[i] = ALPHA;
    else if (c >= '0' and c <= '9')  x[i] = NUMBER;
    else if (c == '`')               x[i] = SYMBOL;
    else if (c == '(')               x[i] = LP;
    else if (c == ')')               x[i] = RP;
    else if (c == '[')               x[i] = LS;
    else if (c == ']')               x[i] = RS;
    else if (c == '{')               x[i] = LC;
    else if (c == '}')               x[i] = RC;
    else if (c == ':')               x[i] = COLON;
    else if (c == '.')               x[i] = DOT;
    else if (c == ';')               x[i] = SEM;
    else if (c > 32 and c < 127)     x[i] = VERB;
    else                             x[i] = UNK;
  }
  if (i==n)                          x[i] = END;
  __syncthreads();
  refine(x, n);
}

// refinements:
// 1. strings start with " and end with a second "
// 2. \ has special meaning inside strings
// 3.  / (note leading space) is a line comment
// 4. `xyz is a symbol
__device__ void refine(int *x, int n) {
  int i = blockIdx.x * blcokDim.x + threadIdx.x;
  if (i<n and i>0) {
    // if inside quotes and current char is \, escape next char
    // if inside quotes
  }
}


int main(int argc, char*argv[]) {
  int *buf;
  hipMallocManaged(&buf, LINE_LEN*sizeof(int)); // buffer for current line tokens

  if (argc == 1) { // repl or stdin
    for (std::string line; std::getline(std::cin, line);) {
      auto sz = line.size();

      scan<<<16,2>>>(buf, line.c_str(), sz);
      hipDeviceSynchronize();

      refine(buf, sz);
      for (auto &c : buf)
        std::cout << c << " ";

      std::cout << std::endl;
    }
    return 0;
  }

  if (argc == 2) { // string eval
    auto line = std::string(argv[1]);
    int sz = line.size();

    scan<<<16,2>>>(buf, line.c_str(), sz);
    hipDeviceSynchronize();

    for (int i=0;i<sz;i++)
      std::cout << buf[i] << " ";

    std::cout << std::endl;

    hipFree(buf);
    return 0;
  }
  return 1; // unknown number of args
}
